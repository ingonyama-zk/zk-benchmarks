#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <benchmark/benchmark.h>

#define CURVE_ID 1
#include "/opt/icicle/icicle/curves/curve_config.cuh"
using namespace curve_config;

#define MAX_THREADS_PER_BLOCK 256

#if CURVE_ID == BN254
const std::string curve = "BN254";
#elif CURVE_ID == BLS12_381
const std::string curve = "BLS12-381";
#elif CURVE_ID == BLS12_377
const std::string curve = "BLS12-377";
#endif

template <typename E, typename S, int N>
__global__ void vectorAdd(S *scalar_vec, E *element_vec, E *result, size_t n_elments)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n_elments)
    {
        const S s = scalar_vec[tid];
        E e = element_vec[tid];
        for (int i = 0; i < N; i++)
            e = e + s;
        result[tid] = e;
    }
}

template <typename E, typename S, int N = 10>
int vector_add(E *vec_b, S *vec_a, E *result, size_t n_elments) // TODO: in place so no need for third result vector
{
    // Set the grid and block dimensions
    int num_blocks = (int)ceil((float)n_elments / MAX_THREADS_PER_BLOCK);
    int threads_per_block = MAX_THREADS_PER_BLOCK;

    // Call the kernel to perform element-wise modular multiplication
    vectorAdd<E, S, N><<<num_blocks, threads_per_block>>>(vec_a, vec_b, result, n_elments);
    return 0;
}

typedef projective_t T;
const unsigned nof_add = 100;
unsigned nof_elements = 1 << 25;  
T *vec_a;
T *vec_b;
T *d_vec_b;
T *d_vec_a, *d_result;
nvmlDevice_t device;

static void BM_add(benchmark::State& state) {  
  for (auto _ : state) {
    vector_add<T, T, nof_add>(d_vec_a, d_vec_b, d_result, nof_elements);
    hipDeviceSynchronize();
  }
  unsigned int power;
  nvmlDeviceGetPowerUsage(device, &power);
  state.counters["PowerUsage"] = int(1.0e-3 * power);
  unsigned int temperature;
  nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature);
  state.counters["Temperature"] = int(temperature);
}

BENCHMARK(BM_add)->MinTime(60.);

int main(int argc, char** argv) {
  hipDeviceReset();
  hipDeviceProp_t deviceProperties;
  int deviceId=0;
  hipGetDeviceProperties(&deviceProperties, deviceId);
  std::string gpu_full_name = deviceProperties.name;
  std::cout << gpu_full_name << std::endl;
  std::string gpu_name = gpu_full_name;
  int gpu_clock_mhz = deviceProperties.clockRate/1000.;

  nvmlInit();
  nvmlDeviceGetHandleByIndex(0, &device);  // for GPU 0

  std::cout << "Setting host data" << std::endl;
  
  vec_a = (T*)malloc(sizeof(T) * nof_elements);
  vec_b = (T*)malloc(sizeof(T) * nof_elements);
  for (unsigned i = 0; i < (1 << 10); i++) {
    vec_a[i] = T::rand_host();
    vec_b[i] = T::rand_host();
  }
  for (unsigned i = 1; i < (nof_elements >> 10); i++) {
    memcpy((void *)(vec_a + (i << 10)), (void *)(vec_a + ((i-1) << 10)), sizeof(T) << 10);
    memcpy((void *)(vec_b + (i << 10)), (void *)(vec_b + ((i-1) << 10)), sizeof(T) << 10);
  }
  // Allocate memory on the device for the input vectors, the output vector, and the modulus
  std::cout << "Moving data to device" << std::endl;
  hipMalloc(&d_vec_a, nof_elements * sizeof(T));
  hipMalloc(&d_vec_b, nof_elements * sizeof(T));
  hipMalloc(&d_result, nof_elements * sizeof(T));

  // Copy the input vectors and the modulus from the host to the device
  hipMemcpy(d_vec_a, vec_a, nof_elements * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(d_vec_b, vec_b, nof_elements * sizeof(T), hipMemcpyHostToDevice);
  std::cout << "Running benchmark" << std::endl;

  // Run all benchmarks 
  ::benchmark::Initialize(&argc, argv);
  if (::benchmark::ReportUnrecognizedArguments(argc, argv)) return 1;
  ::benchmark::AddCustomContext("team", "Ingonyama");
  ::benchmark::AddCustomContext("project", "Icicle");
  ::benchmark::AddCustomContext("runs_on", gpu_name);
  ::benchmark::AddCustomContext("frequency_MHz", std::to_string(gpu_clock_mhz));
  ::benchmark::AddCustomContext("uses", curve);
  ::benchmark::AddCustomContext("comment", "on-device API");
  ::benchmark::AddCustomContext("operation_factor", std::to_string(nof_add));
  ::benchmark::AddCustomContext("vector_size", std::to_string(nof_elements));
  ::benchmark::RunSpecifiedBenchmarks();

  hipFree(d_vec_a);
  hipFree(d_vec_b);
  hipFree(d_result);
  free(vec_a);
  free(vec_b);
}
