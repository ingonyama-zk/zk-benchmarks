// #define CURVE_BN254     1
// #define CURVE_BLS12_381 2
// #define CURVE_BLS12_377 3

// #define CURVE_ID CURVE_BN254

#include <stdio.h>
#include <iostream>
#include <string>
// #include <hip/hip_runtime.h>
#include <nvml.h>
#include <benchmark/benchmark.h>
// #include "icicle/primitives/field.cuh"
// #include "icicle/utils/storage.cuh"
// #include "icicle/primitives/projective.cuh"


// select the curve
#define CURVE_ID 1

// include MSM template
#include "icicle/appUtils/msm/msm.cu"
using namespace curve_config;

#if CURVE_ID == BN254
const std::string curve = "BN254";
#elif CURVE_ID == BLS12_381
const std::string curve = "BLS12-381";
#elif CURVE_ID == BLS12_377
const std::string curve = "BLS12-377";
#endif

const unsigned max_msm_size = 1<<22;
unsigned bucket_factor = 12;

// on-host data
scalar_t* scalars;
affine_t* points;
projective_t result;

// on-device data
scalar_t* scalars_d;
affine_t* points_d;
projective_t* result_d;

nvmlDevice_t device;
hipStream_t stream;

static void BM_msm(benchmark::State& state) {
  const uint32_t msm_size=state.range(0);  
  // Create a device context
  device_context::DeviceContext ctx = {
    stream, // stream
    0,      // device_id
    0,      // mempool
  };
  // Create a MSM configuration
  msm::MSMConfig config = {
    ctx,   // DeviceContext
    0,     // points_size
    1,     // precompute_factor
    0,     // c
    0,     // bitsize
    10,    // large_bucket_factor
    1,     // batch_size
    true, // are_scalars_on_device
    false, // are_scalars_montgomery_form
    true, // are_points_on_device
    false, // are_points_montgomery_form
    true,  // are_results_on_device
    false, // is_big_triangle
    false,  // is_async
  };
  for (auto _ : state) {
    msm::MSM<scalar_t, affine_t, projective_t>(scalars_d, points_d, msm_size, config, result_d);
  }
  unsigned int power;
  nvmlDeviceGetPowerUsage(device, &power);
  state.counters["PowerUsage"] = int(1.0e-3 * power);
  unsigned int temperature;
  nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature);
  state.counters["Temperature"] = int(temperature);
}

BENCHMARK(BM_msm)->MinTime(30.)
  ->Arg(1<<10)
  ->Arg(1<<11)
  ->Arg(1<<12)
  ->Arg(1<<13)
  ->Arg(1<<14)
  ->Arg(1<<15)
  ->Arg(1<<16)
  ->Arg(1<<17)
  ->Arg(1<<18)
  ->Arg(1<<19)
  ->Arg(1<<20)
  ->Arg(1<<21)
  ->Arg(1<<22);

int main(int argc, char** argv) {
  hipDeviceReset();
  hipDeviceProp_t deviceProperties;
  int deviceId=0;
  hipGetDeviceProperties(&deviceProperties, deviceId);
  std::string gpu_full_name = deviceProperties.name;
  std::cout << gpu_full_name << std::endl;
  std::string gpu_name = gpu_full_name;
  int gpu_clock_mhz = deviceProperties.clockRate/1000.;
  nvmlInit();
  nvmlDeviceGetHandleByIndex(0, &device);  // for GPU 0

  std::cout << "Setting host data" << std::endl;
  scalars = (scalar_t*) malloc(sizeof(scalar_t) * max_msm_size);
  points = (affine_t*)malloc(sizeof(affine_t) * max_msm_size);
  scalar_t::RandHostMany(scalars, max_msm_size);
  projective_t::RandHostManyAffine(points, max_msm_size);

  // for (unsigned i = 0; i < max_msm_size; i++) {
  //   points[i] = (i % max_msm_size < 10) ? projective_t::to_affine(projective_t::rand_host()) : points[i - 10];
  //   scalars[i] = scalar_t::rand_host();
  // }

  std::cout << "Moving data to device" << std::endl;
  hipMalloc(&scalars_d, sizeof(scalar_t) * max_msm_size);
  hipMalloc(&points_d, sizeof(affine_t) * max_msm_size);
  hipMalloc(&result_d, sizeof(projective_t));
  hipMemcpy(scalars_d, scalars, sizeof(scalar_t) * max_msm_size, hipMemcpyHostToDevice);
  hipMemcpy(points_d, points, sizeof(affine_t) * max_msm_size, hipMemcpyHostToDevice);


  std::cout << "Running benchmark" << std::endl;
  hipStreamCreate(&stream);

  // Run all benchmarks 
  ::benchmark::Initialize(&argc, argv);
  if (::benchmark::ReportUnrecognizedArguments(argc, argv)) return 1;
  ::benchmark::AddCustomContext("team", "Ingonyama");
  ::benchmark::AddCustomContext("project", "Icicle");
  ::benchmark::AddCustomContext("runs_on", gpu_name);
  ::benchmark::AddCustomContext("frequency_MHz", std::to_string(gpu_clock_mhz));
  ::benchmark::AddCustomContext("uses", curve);
  ::benchmark::AddCustomContext("comment", "on-device API");
  ::benchmark::AddCustomContext("coefficient_C", std::to_string(bucket_factor));
  ::benchmark::RunSpecifiedBenchmarks();

  std::cout << "Cleaning up" << std::endl;
  hipFree(scalars_d);
  hipFree(points_d);
  hipFree(result_d);
  free(scalars);
  free(points);
  return 0;
}
