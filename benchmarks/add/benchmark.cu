#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <benchmark/benchmark.h>
#include "icicle/primitives/field.cuh"
#include "icicle/utils/storage.cuh"
#include "icicle/primitives/projective.cuh"
#include "icicle/curves/bn254/curve_config.cuh"
#include "ve_mod_mult.cuh"
 
using namespace BN254;

typedef projective_t T;
const unsigned nof_add = 100;
unsigned nof_elements;
T *vec_a;
T *vec_b;
T *d_vec_b;
T *d_vec_a, *d_result;
nvmlDevice_t device;

static void BM_add(benchmark::State& state) {  
  for (auto _ : state) {
    vector_add<T, T, nof_add>(d_vec_a, d_vec_b, d_result, nof_elements);
    hipDeviceSynchronize();
  }
  unsigned int power;
  nvmlDeviceGetPowerUsage(device, &power);
  state.counters["PowerUsage"] = int(1.0e-3 * power);
  unsigned int temperature;
  nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature);
  state.counters["Temperature"] = int(temperature);
}

BENCHMARK(BM_add)->MinTime(30.);

int main(int argc, char** argv) {
  hipDeviceReset();
  hipDeviceProp_t deviceProperties;
  int deviceId=0;
  hipGetDeviceProperties(&deviceProperties, deviceId);
  std::string gpu_full_name = deviceProperties.name;
  std::cout << gpu_full_name << std::endl;
  std::string gpu_name;
  if (gpu_full_name.find("3090") != std::string::npos) {
    gpu_name = "RTX 3090";
  } else if (gpu_name.find("4090") != std::string::npos) {
    gpu_name = "RTX 4090";
  } else {
        std::cout << "unrecognized GPU" << std::endl;
  }
  int gpu_clock_mhz = deviceProperties.clockRate/1000.;

  nvmlInit();
  nvmlDeviceGetHandleByIndex(0, &device);  // for GPU 0

  std::cout << "Setting host data" << std::endl;
  nof_elements = 1 << 25;  
  vec_a = (T*)malloc(sizeof(T) * nof_elements);
  vec_b = (T*)malloc(sizeof(T) * nof_elements);
  for (unsigned i = 0; i < (1 << 10); i++) {
    vec_a[i] = T::rand_host();
    vec_b[i] = T::rand_host();
  }
  for (unsigned i = 1; i < (nof_elements >> 10); i++) {
    memcpy((void *)(vec_a + (i << 10)), (void *)(vec_a + ((i-1) << 10)), sizeof(T) << 10);
    memcpy((void *)(vec_b + (i << 10)), (void *)(vec_b + ((i-1) << 10)), sizeof(T) << 10);
  }
  // Allocate memory on the device for the input vectors, the output vector, and the modulus
  std::cout << "Moving data to device" << std::endl;
  hipMalloc(&d_vec_a, nof_elements * sizeof(T));
  hipMalloc(&d_vec_b, nof_elements * sizeof(T));
  hipMalloc(&d_result, nof_elements * sizeof(T));

  // Copy the input vectors and the modulus from the host to the device
  hipMemcpy(d_vec_a, vec_a, nof_elements * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(d_vec_b, vec_b, nof_elements * sizeof(T), hipMemcpyHostToDevice);
  std::cout << "Running benchmark" << std::endl;

  // Run all benchmarks
  ::benchmark::Initialize(&argc, argv);
  if (::benchmark::ReportUnrecognizedArguments(argc, argv)) return 1;
  ::benchmark::AddCustomContext("project", "ICICLE");
  ::benchmark::AddCustomContext("runs_on", gpu_name);
  ::benchmark::AddCustomContext("frequency_MHz", std::to_string(gpu_clock_mhz));
  ::benchmark::AddCustomContext("uses", "BN254");
  ::benchmark::AddCustomContext("comment", "on-device API");
  ::benchmark::RunSpecifiedBenchmarks();


  hipFree(d_vec_a);
  hipFree(d_vec_b);
  hipFree(d_result);
  free(vec_a);
  free(vec_b);
}
