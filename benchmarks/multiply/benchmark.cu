#include "hip/hip_runtime.h"
#define CURVE_BN254     1
#define CURVE_BLS12_381 2
#define CURVE_BLS12_377 3

#define CURVE CURVE_BLS12_377

#include <stdio.h>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include </opt/benchmark/include/benchmark/benchmark.h>
#include "/icicle/icicle/primitives/field.cuh"

#if CURVE == CURVE_BN254

#include "/icicle/icicle/curves/bn254/curve_config.cuh"    
using namespace BN254;
const std::string curve = "BN254";

#elif CURVE == CURVE_BLS12_381

#include "/icicle/icicle/curves/bls12_381/curve_config.cuh"
using namespace BLS12_381;
const std::string curve = "BLS12-381";

#elif CURVE == CURVE_BLS12_377

#include "/icicle/icicle/curves/bls12_377/curve_config.cuh"
using namespace BLS12_377;
const std::string curve = "BLS12-377";
    
#endif


#include "/icicle/icicle/appUtils/vector_manipulation/ve_mod_mult.cuh"

template <typename S, int N>
__global__ void vectorMult(S *vec_a, S *vec_b, S *vec_r, size_t n_elments)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n_elments)
    {
        const S b = vec_b[tid];
        S r = vec_a[tid];
        // #pragma unroll
        for (int i = 0; i < N; i++)
            r = r * b;
        vec_r[tid] = r;
    }
}

template <typename S, int N = 10>
int vector_mult(S *vec_b, S *vec_a, S *vec_result, size_t n_elments)
{
    // Set the grid and block dimensions
    int num_blocks = (int)ceil((float)n_elments / MAX_THREADS_PER_BLOCK);
    int threads_per_block = MAX_THREADS_PER_BLOCK;

    // Call the kernel to perform element-wise modular multiplication
    vectorMult<S, N><<<num_blocks, threads_per_block>>>(vec_a, vec_b, vec_result, n_elments);
    return 0;
}

// typedef scalar_t S;
typedef point_field_t S;

const unsigned nof_mult = 100;
unsigned nof_elements = 1 << 25;  
S *vec_a;
S *vec_b;
S *d_vec_b;
S *d_vec_a, *d_result;
nvmlDevice_t device;

static void BM_mult(benchmark::State& state) {  
  for (auto _ : state) {
    vector_mult<S, nof_mult>(d_vec_a, d_vec_b, d_result, nof_elements);
    hipDeviceSynchronize();
  }
  unsigned int power;
  nvmlDeviceGetPowerUsage(device, &power);
  state.counters["PowerUsage"] = int(1.0e-3 * power);
  unsigned int temperature;
  nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature);
  state.counters["Temperature"] = int(temperature);
}

BENCHMARK(BM_mult)->MinTime(60.);

int main(int argc, char** argv) {
  hipDeviceReset();
  hipDeviceProp_t deviceProperties;
  int deviceId=0;
  hipGetDeviceProperties(&deviceProperties, deviceId);
  std::string gpu_full_name = deviceProperties.name;
  std::cout << gpu_full_name << std::endl;
  std::string gpu_name = gpu_full_name;
  int gpu_clock_mhz = deviceProperties.clockRate/1000.;

  nvmlInit();
  nvmlDeviceGetHandleByIndex(0, &device);  // for GPU 0

  std::cout << "Setting host data" << std::endl;
  
  vec_a = (S*)malloc(sizeof(S) * nof_elements);
  vec_b = (S*)malloc(sizeof(S) * nof_elements);
  for (unsigned i = 0; i < (1 << 10); i++) {
    vec_a[i] = S::rand_host();
    vec_b[i] = S::rand_host();
  }
  for (unsigned i = 1; i < (nof_elements >> 10); i++) {
    memcpy((void *)(vec_a + (i << 10)), (void *)(vec_a + ((i-1) << 10)), sizeof(S) << 10);
    memcpy((void *)(vec_b + (i << 10)), (void *)(vec_b + ((i-1) << 10)), sizeof(S) << 10);
  }
  // Allocate memory on the device for the input vectors, the output vector, and the modulus
  std::cout << "Moving data to device" << std::endl;
  hipMalloc(&d_vec_a, nof_elements * sizeof(S));
  hipMalloc(&d_vec_b, nof_elements * sizeof(S));
  hipMalloc(&d_result, nof_elements * sizeof(S));

  // Copy the input vectors and the modulus from the host to the device
  hipMemcpy(d_vec_a, vec_a, nof_elements * sizeof(S), hipMemcpyHostToDevice);
  hipMemcpy(d_vec_b, vec_b, nof_elements * sizeof(S), hipMemcpyHostToDevice);
  std::cout << "Running benchmark" << std::endl;

  // Run all benchmarks 
  ::benchmark::Initialize(&argc, argv);
  if (::benchmark::ReportUnrecognizedArguments(argc, argv)) return 1;
  ::benchmark::AddCustomContext("team", "Ingonyama");
  ::benchmark::AddCustomContext("project", "Icicle");
  ::benchmark::AddCustomContext("runs_on", gpu_name);
  ::benchmark::AddCustomContext("frequency_MHz", std::to_string(gpu_clock_mhz));
  ::benchmark::AddCustomContext("uses", curve);
  ::benchmark::AddCustomContext("comment", "on-device API");
  ::benchmark::AddCustomContext("operation_factor", std::to_string(nof_mult));
  ::benchmark::AddCustomContext("vector_size", std::to_string(nof_elements));
  ::benchmark::RunSpecifiedBenchmarks();

  hipFree(d_vec_a);
  hipFree(d_vec_b);
  hipFree(d_result);
  free(vec_a);
  free(vec_b);
}
